#include "hip/hip_runtime.h"
//Includes

#include "util.cuh" //general utilities

#include <glob.h> //pathname pattern matching types

//Functions

//set log file and open it
void logger::set_file(const std::string &pathstr) //file path string
{
  logger &sin = get_instance(); //singleton instance
  if (sin.log_f.is_open()){ sin.log_f.close();}
  sin.log_f.open(pathstr,std::ios::app);
  sin.log_f.close();
  sin.log_f.open(pathstr,std::ios::in|std::ios::ate);
  if (sin.log_f.is_open()){ sin.w_f = true;}
  else{ sin.w_f = false;}
}

//log message with timestamp
void logger::record(const std::string &msg) //message
{
  logger &sin = get_instance(); //singleton instance
  time_t now = time(nullptr); //current time
  tm *now_info = localtime(&now); //curent time information
  char timestr[21]; //timestamp C-style string
  strftime(timestr,21,"[%d/%m/%y %H:%M:%S] ",now_info);
  if (sin.w_f)
  {
    sin.log_f<<timestr<<msg<<std::endl;
  }
  std::cout<<timestr<<msg<<std::endl;
}

//show progress percentage
void logger::show_prog_pc(float prog_pc) //progress percentage
{
  logger &sin = get_instance(); //singleton instance
  if (sin.w_f)
  {
    sin.log_f<<"progress: "<<cnfs(prog_pc,5,'0',1)<<"%";
    sin.log_f.seekp(-16,std::ios::cur);
  }
  std::cout<<"progress: "<<cnfs(prog_pc,5,'0',1)<<"%";
  std::cout<<"\r"; std::cout.flush();
}

//basic logger constructor
logger::logger() {}

//basic logger destructor
logger::~logger()
{
  log_f.close();
}

//return singleton instance
logger &logger::get_instance()
{
  static logger sin; //singleton instance
  return sin;
}

//generic exception type constructor
error::error(const std::string &msg) //error message
  : std::runtime_error(msg) {}

//check for errors in cuda runtime API call
void cuda_check(hipError_t rtn_val) //cuda runtime API call return value
{
  if (rtn_val!=hipSuccess)
  {
    std::string msg = "cuda: "; //error message
    msg += hipGetErrorString(rtn_val);
    throw error(msg);
  }
}

//count files matching pattern
int glob_count(const std::string &pathpat) //file path pattern
{
  glob_t glob_sr; //glob search result
  int rtn_val = glob(pathpat.c_str(),0,nullptr,&glob_sr); //return value
  if (rtn_val!=0)
  {
    globfree(&glob_sr);
    if (rtn_val==GLOB_NOMATCH){ return 0;}
    else{ throw error("unable to find matches of "+pathpat);}
  }
  else
  {
    globfree(&glob_sr);
    return glob_sr.gl_pathc;
  }
}
