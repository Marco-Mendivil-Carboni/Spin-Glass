#include "hip/hip_runtime.h"
//Includes

#include "eamsim.cuh" //EA model simulation

//Device Functions

//initialize probability lookup table
inline __device__ void init_prob(
  uint64_t s_prob[NREP][PTABW], //shared probability lookup table
  const int i_bt) //block thread index
{
  //initialize all entries to 1
  if (i_bt<PTABW)
  {
    for (int i_r = 0; i_r<NREP; ++i_r) //replica index
    {
      s_prob[i_r][i_bt] = UINT_MAX;
    }
  }
  __syncthreads();
}

//compute probability lookup table
inline __device__ void compute_prob(
  uint64_t s_prob[NREP][PTABW], //shared probability lookup table
  float *s_rep_beta, //shared replica beta array
  const float H, //external magnetic field
  const int i_bt) //block thread index
{
  //compute all possible probabilities
  if (i_bt<NPROB)
  {
    for (int i_r = 0; i_r<NREP; ++i_r) //replica index
    {
      float energy = -(i_bt-6-H-((1-2*H)*(i_bt&1))); //spin energy
      s_prob[i_r][i_bt] = expf(s_rep_beta[i_r]*2*energy)*UINT_MAX;
    }
  }
  __syncthreads();
}

//shuffle lattice temperature replicas
inline __device__ void shuffle(
  int *s_rep_idx, //shared replica index array
  float *s_rep_beta, //shared replica beta array
  float *s_tot_sum_e, //shared total energy sum array
  prng *prngs, //PRNG state array
  const int i_bt, //block thread index
  const int i_gt, //grid thread index
  bool mode) //shuffle mode
{
  //declare auxiliary variables
  int i_0; //1st array index
  int i_1; //2nd array index
  int max_i_bt; //maximum block thread index
  __shared__ int s_rai[NREP]; //shared rearranged array index array

  //write shared rearranged array index array
  if (i_bt<NREP){ s_rai[s_rep_idx[i_bt]] = i_bt;}
  __syncthreads ();

  if (mode==0) //consider even pairs of temperature replicas
  {
    i_0 = s_rai[(i_bt<<1)+0]; i_1 = s_rai[(i_bt<<1)+1]; max_i_bt = NREP/2;
  }
  else //consider odd pairs of temperature replicas
  {
    i_0 = s_rai[(i_bt<<1)+1]; i_1 = s_rai[(i_bt<<1)+2]; max_i_bt = NREP/2-1;
  }

  if (i_bt<max_i_bt) //shuffle pair of temperature replicas
  {
    //generate random number
    float ran = hiprand_uniform(&prngs[i_gt]); //random number in (0,1]

    //compute shuffle probability
    float beta_diff = s_rep_beta[i_0]-s_rep_beta[i_1]; //beta difference
    float energy_diff = s_tot_sum_e[i_0]-s_tot_sum_e[i_1]; //energy difference
    float prob = expf(beta_diff*energy_diff); //shuffle probability

    if (ran<prob) //accept shuffle
    {
      int tmp_idx = s_rep_idx[i_0]; //temporary index
      s_rep_idx[i_0] = s_rep_idx[i_1]; s_rep_idx[i_1] = tmp_idx;
      float tmp_beta = s_rep_beta[i_0]; //temporary beta
      s_rep_beta[i_0] = s_rep_beta[i_1]; s_rep_beta[i_1] = tmp_beta;
    }
  }
  __syncthreads();
}

//perform skewed sequential sum reduction
inline __device__ void sum_reduce(
  float *s_tot_sum, //shared total sum array
  short s_aux_sum[NREP][NTPB], //shared auxiliary sums array
  const int i_bt) //block thread index
{
  //sum auxiliary sums for each temperature replica
  if (i_bt<NREP)
  {
    int sum = 0; //sum of auxiliary sums
    for (int i_sl = 0; i_sl<NTPB; ++i_sl) //skewed loop index
    {
      sum += s_aux_sum[i_bt][(i_sl+i_bt)%NTPB];
    }
    s_tot_sum[i_bt] = sum;
  }
  __syncthreads();
}

//perform Monte Carlo steps
inline __device__ void perform_MC_steps(
  uint32_t *slattice, //shuffled lattice array
  float *s_rep_beta, //shared replica beta array
  const float H, //external magnetic field
  prng *prngs, //PRNG state array
  int n_steps) //number of Monte Carlo steps
{
  //calculate indexes
  const int i_gb = blockIdx.x; //grid block index
  const int i_bt = //block thread index
    CBDIM.x*CBDIM.y*threadIdx.z+CBDIM.x*threadIdx.y+threadIdx.x;
  const int i_gt = CBDIM.x*CBDIM.y*CBDIM.z*i_gb+i_bt; //grid thread index

  //declare auxiliary variables
  __shared__ uint32_t s_slattice[L][L][L]; //shared shuffled lattice array
  __shared__ uint64_t s_prob[NREP][PTABW]; //shared probability lookup table

  //initilize probability lookup table
  init_prob(s_prob,i_bt);

  //compute probability lookup table
  compute_prob(s_prob,s_rep_beta,H,i_bt);

  //write shared shuffled lattice array
  int xt = (L/2*(threadIdx.y&1))+threadIdx.x; //total x index
  int yt = (L/2*(threadIdx.z&1))+(threadIdx.y>>1); //total y index
  for (int zt = 0; zt<L; ++zt) //total z index
  {
    s_slattice[zt][yt][xt] = slattice[N*i_gb+L*L*zt+i_bt];
  }
  __syncthreads();

  //perform all Monte Carlo steps
  for (int step = 0; step<n_steps; ++step) //Monte Carlo step index
  {
    //perform both phases of each update
    for (int phase = 0; phase<2; ++phase) //update phase index
    {
      //calculate shared shuffled lattice indexes
      int xs = (threadIdx.z&1)^(threadIdx.y&1)^phase; //starting x index
      int xc = (threadIdx.x<<1)+xs; //centered x index
      int xr = (xc+L-1)%L; //retarded x index
      int xa = (xc+1)%L; //advanced x index
      int yc = threadIdx.y; //centered y index
      int yr = (yc+L-1)%L; //retarded y index
      int ya = (yc+1)%L; //advanced y index
      for (int oz = 0; oz<L; oz += CBDIM.z) //z index offset
      {
        int zc = oz+threadIdx.z; //centered z index
        int zr = (zc+L-1)%L; //retarded z index
        int za = (zc+1)%L; //advanced z index

        //compute interactions with first neighbours
        uint32_t cmspin = s_slattice[zc][yc][xc]; //centered multispin
        uint32_t int_0 = s_slattice[zc][yc][xr]; //interaction 0 (left)
        uint32_t int_1 = s_slattice[zc][yc][xa]; //interaction 1 (right)
        uint32_t int_2 = s_slattice[zc][yr][xc]; //interaction 2 (down)
        uint32_t int_3 = s_slattice[zc][ya][xc]; //interaction 3 (up)
        uint32_t int_4 = s_slattice[zr][yc][xc]; //interaction 4 (back)
        uint32_t int_5 = s_slattice[za][yc][xc]; //interaction 5 (front)
        int_0 = (MASKAB*((cmspin>>(SHIFTSJ+0))&1))^int_0^cmspin;
        int_1 = (MASKAB*((cmspin>>(SHIFTSJ+1))&1))^int_1^cmspin;
        int_2 = (MASKAB*((cmspin>>(SHIFTSJ+2))&1))^int_2^cmspin;
        int_3 = (MASKAB*((cmspin>>(SHIFTSJ+3))&1))^int_3^cmspin;
        int_4 = (MASKAB*((cmspin>>(SHIFTSJ+4))&1))^int_4^cmspin;
        int_5 = (MASKAB*((cmspin>>(SHIFTSJ+5))&1))^int_5^cmspin;

        //flip every spin in the multispin
        for (int i_ss = 0; i_ss<NSPS; ++i_ss) //segment spin index
        {
          //compute energy-spin index
          uint32_t es_idx = //energy-spin index
            ((int_0>>i_ss)&MASKSS)+
            ((int_1>>i_ss)&MASKSS)+
            ((int_2>>i_ss)&MASKSS)+
            ((int_3>>i_ss)&MASKSS)+
            ((int_4>>i_ss)&MASKSS)+
            ((int_5>>i_ss)&MASKSS);
          es_idx = (es_idx<<1)+((cmspin>>i_ss)&MASKSS);

          //compute spin flips
          uint32_t flip = 0; //spin flips
          for (int shift = 0; shift<SHIFTMS; shift += NSPS) //segment shift
          {
            //generate random unsigned integer
            uint64_t ran = hiprand(&prngs[i_gt]); //random unsigned integer

            //compute flip probability
            uint64_t prob = //flip probability
              s_prob[shift+i_ss][(es_idx>>shift)&MASKES];

            //update spin flips
            flip |= (ran<prob)<<shift;
          }

          //flip spins
          cmspin ^= (flip<<i_ss);
        }

        //update shared shuffled lattice array
        s_slattice[zc][yc][xc] = cmspin;
      }
      __syncthreads();
    }
  }

  //write shuffled lattice array
  for (int zt = 0; zt<L; ++zt) //total z index
  {
    slattice[N*i_gb+L*L*zt+i_bt] = s_slattice[zt][yt][xt];
  }
  __syncthreads();
}

//perform Parallel Tempering shuffle
inline __device__ void perform_PT_shuffle(
  uint32_t *slattice, //shuffled lattice array
  int *s_rep_idx, //shared replica index array
  float *s_rep_beta, //shared replica beta array
  float *s_tot_sum_e, //shared total energy sum array
  float *s_tot_sum_m, //shared total magnetization sum array
  const float H, //external magnetic field
  prng *prngs, //PRNG state array
  bool mode) //shuffle mode
{
  //calculate indexes
  const int i_gb = blockIdx.x; //grid block index
  const int i_bt = //block thread index
    CBDIM.x*CBDIM.y*threadIdx.z+CBDIM.x*threadIdx.y+threadIdx.x;
  const int i_gt = CBDIM.x*CBDIM.y*CBDIM.z*i_gb+i_bt; //grid thread index

  //declare auxiliary variables
  __shared__ uint32_t s_slattice[L][L][L]; //shared shuffled lattice array
  __shared__ short s_aux_sum[NREP][NTPB]; //shared auxiliary sums array

  //write shared shuffled lattice array
  int xt = (L/2*(threadIdx.y&1))+threadIdx.x; //total x index
  int yt = (L/2*(threadIdx.z&1))+(threadIdx.y>>1); //total y index
  for (int zt = 0; zt<L; ++zt) //total z index
  {
    s_slattice[zt][yt][xt] = slattice[N*i_gb+L*L*zt+i_bt];
  }
  __syncthreads();

  //initialize shared auxiliary sums array to 0
  for (int i_r = 0; i_r<NREP; ++i_r) //replica index
  {
    s_aux_sum[i_r][i_bt] = 0;
  }
  __syncthreads();

  //calculate shared shuffled lattice indexes
  int xs = (threadIdx.z&1)^(threadIdx.y&1); //starting x index
  int xc = (threadIdx.x<<1)+xs; //centered x index
  int xr = (xc+L-1)%L; //retarded x index
  int xa = (xc+1)%L; //advanced x index
  int xm = xc-2*xs+1; //matching x index
  int yc = threadIdx.y; //centered y index
  int yr = (yc+L-1)%L; //retarded y index
  int ya = (yc+1)%L; //advanced y index
  for (int oz = 0; oz<L; oz += CBDIM.z) //z index offset
  {
    int zc = oz+threadIdx.z; //centered z index
    int zr = (zc+L-1)%L; //retarded z index
    int za = (zc+1)%L; //advanced z index

    //compute interactions with first neighbours
    uint32_t cmspin = s_slattice[zc][yc][xc]; //centered multispin
    uint32_t int_0 = s_slattice[zc][yc][xr]; //interaction 0 (left)
    uint32_t int_1 = s_slattice[zc][yc][xa]; //interaction 1 (right)
    uint32_t int_2 = s_slattice[zc][yr][xc]; //interaction 2 (down)
    uint32_t int_3 = s_slattice[zc][ya][xc]; //interaction 3 (up)
    uint32_t int_4 = s_slattice[zr][yc][xc]; //interaction 4 (back)
    uint32_t int_5 = s_slattice[za][yc][xc]; //interaction 5 (front)
    int_0 = (MASKAB*((cmspin>>(SHIFTSJ+0))&1))^int_0^cmspin;
    int_1 = (MASKAB*((cmspin>>(SHIFTSJ+1))&1))^int_1^cmspin;
    int_2 = (MASKAB*((cmspin>>(SHIFTSJ+2))&1))^int_2^cmspin;
    int_3 = (MASKAB*((cmspin>>(SHIFTSJ+3))&1))^int_3^cmspin;
    int_4 = (MASKAB*((cmspin>>(SHIFTSJ+4))&1))^int_4^cmspin;
    int_5 = (MASKAB*((cmspin>>(SHIFTSJ+5))&1))^int_5^cmspin;

    //add energy indexes to shared auxiliary sums array
    for (int i_ss = 0; i_ss<NSPS; ++i_ss) //segment spin index
    {
      uint32_t e_idx = //energy index
        ((int_0>>i_ss)&MASKSS)+
        ((int_1>>i_ss)&MASKSS)+
        ((int_2>>i_ss)&MASKSS)+
        ((int_3>>i_ss)&MASKSS)+
        ((int_4>>i_ss)&MASKSS)+
        ((int_5>>i_ss)&MASKSS);
      for (int shift = 0; shift<SHIFTMS; shift += NSPS) //segment shift
      {
        s_aux_sum[shift+i_ss][i_bt] += (e_idx>>shift)&MASKES;
      }
    }
  }
  __syncthreads();

  //perform sum reduction of energy indexes
  sum_reduce(s_tot_sum_e,s_aux_sum,i_bt);

  //reset shared auxiliary sums array to 0
  for (int i_r = 0; i_r<NREP; ++i_r) //replica index
  {
    s_aux_sum[i_r][i_bt] = 0;
  }
  __syncthreads();

  //calculate shared shuffled lattice indexes
  for (int oz = 0; oz<L; oz += CBDIM.z) //z index offset
  {
    int zc = oz+threadIdx.z; //centered z index

    //read lattice multispins
    uint32_t cmpsin = s_slattice[zc][yc][xc]; //centered multispin
    uint32_t mmpsin = s_slattice[zc][yc][xm]; //matching multispin

    //add spin indexes to shared auxiliary sums array
    for (int i_ss = 0; i_ss<NSPS; ++i_ss) //segment spin index
    {
      for (int shift = 0; shift<SHIFTMS; shift += NSPS) //segment shift
      {
        int i_r = shift+i_ss; //replica index
        s_aux_sum[i_r][i_bt] += ((cmpsin>>i_r)&1)+((mmpsin>>i_r)&1);
      }
    }
  }
  __syncthreads();

  //perform sum reduction of spin indexes
  sum_reduce(s_tot_sum_m,s_aux_sum,i_bt);

  //shift both energies to their physical value and add them
  if (i_bt<NREP)
  {
    s_tot_sum_e[i_bt] = 2*s_tot_sum_e[i_bt]-6*(N/2);
    s_tot_sum_m[i_bt] = 2*s_tot_sum_m[i_bt]-1*N;
    s_tot_sum_e[i_bt] = -(s_tot_sum_e[i_bt]+H*s_tot_sum_m[i_bt]);
  }
  __syncthreads();

  //shuffle lattice temperature replicas
  shuffle(s_rep_idx,s_rep_beta,s_tot_sum_e,prngs,i_bt,i_gt,mode);
}

//Global Functions

//initialize PRNG state array
__global__ void init_prng(
  prng *prngs, //PRNG state array
  int pseed) //PRNG seed
{
  //calculate grid thread index
  const int i_gt = NTPB*blockIdx.x+threadIdx.x; //grid thread index

  //initialize PRNG state
  hiprand_init(pseed,i_gt,0,&prngs[i_gt]);
}

//run simulation section between measurements
__global__ void run_simulation_section(
  uint32_t *slattice, //shuffled lattice array
  obs_s *obs, //observables array
  ib_s *repib, //replica index-beta array
  const float H, //external magnetic field
  prng *prngs) //PRNG state array
{
  //calculate indexes
  const int i_gb = blockIdx.x; //grid block index
  const int i_bt = //block thread index
    CBDIM.x*CBDIM.y*threadIdx.z+CBDIM.x*threadIdx.y+threadIdx.x;

  //declare auxiliary variables
  __shared__ int s_rep_idx[NREP]; //shared replica index array
  __shared__ float s_rep_beta[NREP]; //shared replica beta array
  __shared__ int s_prev_rep_idx[NREP]; //shared previous replica index array
  __shared__ float s_tot_sum_e[NREP]; //shared total energy sum array
  __shared__ float s_tot_sum_m[NREP]; //shared total magnetization sum array

  //write shared replica index and beta arrays
  if (i_bt<NREP)
  {
    s_rep_idx[i_bt] = repib[NREP*i_gb+i_bt].idx;
    s_rep_beta[i_bt] = repib[NREP*i_gb+i_bt].beta;
  }
  __syncthreads();

  //perform Parallel Tempering shuffles and Monte Carlo steps
  for (int step = 0; step<SBMEAS; step += SBSHFL) //Monte Carlo step index
  {
    bool mode = (step/SBSHFL)&1; //shuffle mode
    if (i_bt<NREP){ s_prev_rep_idx[i_bt] = s_rep_idx[i_bt];}
    perform_PT_shuffle(slattice,s_rep_idx,s_rep_beta,s_tot_sum_e,s_tot_sum_m,H,
      prngs,mode);
    perform_MC_steps(slattice,s_rep_beta,H,prngs,SBSHFL);
  }

  //update replica index-beta array
  if (i_bt<NREP)
  {
    repib[NREP*i_gb+i_bt].idx = s_rep_idx[i_bt];
    repib[NREP*i_gb+i_bt].beta = s_rep_beta[i_bt];
  }

  //write observables array
  if (i_bt<NREP)
  {
    int i_d = i_gb/NCP; //disorder index
    int i_r = s_prev_rep_idx[i_bt]; //replica index
    int i_c = i_gb%NCP; //copy index
    obs[NREP*i_d+i_r].e[i_c] = s_tot_sum_e[i_bt]/N;
    obs[NREP*i_d+i_r].m[i_c] = s_tot_sum_m[i_bt]/N;
  }
}

//rearrange lattice temperature replicas
__global__ void rearrange(
  uint32_t *lattice, //lattice array
  ib_s *repib, //replica index-beta array
  uint32_t *slattice) //shuffled lattice array
{
  //calculate indexes
  const int i_gb = blockIdx.x; //grid block index
  const int i_bt = threadIdx.x; //block thread index

  //declare auxiliary variables
  uint32_t smspin; //shuffled multispin
  uint32_t rmspin; //rearranged multispin
  __shared__ int s_rep_idx[NREP]; //shared replica index array

  //write shared replica index array
  if (i_bt<NREP){ s_rep_idx[i_bt] = repib[NREP*i_gb+i_bt].idx;}
  __syncthreads();

  //update lattice array
  for (int i_s = i_bt; i_s<N; i_s += NTPB) //site index
  {
    smspin = slattice[N*i_gb+i_s];
    rmspin = 0;
    for (int i_r = 0; i_r<NREP; ++i_r) //replica index
    {
      rmspin |= ((smspin>>i_r)&1)<<s_rep_idx[i_r];
    }
    lattice[N*i_gb+i_s] = (lattice[N*i_gb+i_s]&MASKAJ)|rmspin;
  }
}

//compute overlap
__global__ void compute_q(
  obs_s *obs, //observables array
  uint32_t *lattice) //lattice array
{
  //calculate indexes
  const int i_gb = blockIdx.x; //grid block index
  const int i_bt = threadIdx.x; //block thread index

  //declare auxiliary variables
  int l_s[NCP]; //local spin array
  float q_0; //overlap value 0
  float q_1_r[3]; //Re overlap values 1
  float q_1_i[3]; //Im overlap values 1

  //compute overlap values for each temperature replica
  if (i_bt<NREP)
  {
    //initialize overlap values
    q_0 = 0.0;
    for(int i_q = 0; i_q<3; ++i_q) //overlap value index
    {
      q_1_r[i_q] = 0.0;
      q_1_i[i_q] = 0.0;
    }

    //iterate over all sites
    for (int i_s = 0; i_s<N; ++i_s) //site index
    {
      //compute local spin array
      for (int i_c = 0; i_c<NCP; ++i_c) //copy index
      {
        int sbit = (lattice[N*(NCP*i_gb+i_c)+i_s]>>i_bt)&1; //spin bit
        l_s[i_c] = 2*sbit-1;
      }

      //calculate local overlap, wave vector norm and position
      float l_q = l_s[0]*l_s[1]; //local overlap
      float k = 2*M_PI/L; //wave vector norm
      float x = i_s%L; //x position
      float y = (i_s/L)%L; //y position
      float z = (i_s/L)/L; //z position

      //compute overlap values
      q_0 += l_q;
      q_1_r[0] += l_q*cosf(k*x);
      q_1_i[0] += l_q*sinf(k*x);
      q_1_r[1] += l_q*cosf(k*y);
      q_1_i[1] += l_q*sinf(k*y);
      q_1_r[2] += l_q*cosf(k*z);
      q_1_i[2] += l_q*sinf(k*z);
    }

    //write observables array
    obs[NREP*i_gb+i_bt].q_0 = q_0/N;
    for(int i_q = 0; i_q<3; ++i_q) //overlap value index
    {
      obs[NREP*i_gb+i_bt].q_1_r[i_q] = q_1_r[i_q]/N;
      obs[NREP*i_gb+i_bt].q_1_i[i_q] = q_1_i[i_q]/N;
    }
  }
}

//Host Functions

//EA model simulation constructor
eamsim::eamsim(float H) //external magnetic field
  : H {H}
{
  //check parameters
  if (!(0.0<=H&&H<=4.0)){ throw error("H out of range");}
  logger::record("H = "+cnfs(H,6,'0',4));

  //allocate device memory
  cuda_check(hipMalloc(&repib,NREP*NL*sizeof(ib_s)));
  cuda_check(hipMalloc(&prngs,NTPB*NL*sizeof(prng)));
  cuda_check(hipMalloc(&lattice,N*NL*sizeof(uint32_t)));
  cuda_check(hipMalloc(&slattice,N*NL*sizeof(uint32_t)));
  cuda_check(hipMalloc(&obs,NREP*NDIS*sizeof(obs_s)));

  //allocate host memory
  cuda_check(hipHostMalloc(&repib_h,NREP*NL*sizeof(ib_s)));
  cuda_check(hipHostMalloc(&lattice_h,N*NL*sizeof(uint32_t)));
  cuda_check(hipHostMalloc(&obs_h,NREP*NDIS*sizeof(obs_s)));

  //initialize replica index-beta array
  init_repib();

  //initialize PRNG state array
  init_prng<<<NL,NTPB>>>(prngs,time(nullptr));

  //record success message
  logger::record("eamsim initialized");
}

//EA model simulation destructor
eamsim::~eamsim()
{
  //deallocate device memory
  cuda_check(hipFree(repib));
  cuda_check(hipFree(prngs));
  cuda_check(hipFree(lattice));
  cuda_check(hipFree(slattice));
  cuda_check(hipFree(obs));

  //deallocate host memory
  cuda_check(hipHostFree(repib_h));
  cuda_check(hipHostFree(lattice_h));
  cuda_check(hipHostFree(obs_h));
}

//initialize lattice multispins
void init_multispins(
  hiprandGenerator_t gen, //host PRNG
  uint32_t *lattice_h) //lattice host array
{
  //declare auxiliary variables
  uint32_t ranmspin; //random multispin

  //set random lattice multispins
  for (int i_s = 0; i_s<N; ++i_s) //site index
  {
    hiprandGenerate(gen,&ranmspin,1);
    lattice_h[i_s] = (lattice_h[i_s]&MASKAJ)|(ranmspin&MASKAS);
  }
}

//initialize lattice coupling constants
void init_coupling_constants(
  hiprandGenerator_t gen, //host PRNG
  uint32_t *lattice_h) //lattice host array
{
  //declare auxiliary variables
  uint32_t ran[3]; //random numbers
  uint32_t Jx[N]; //x coupling constants
  uint32_t Jy[N]; //y coupling constants
  uint32_t Jz[N]; //z coupling constants

  //choose random coupling constants
  for (int i_s = 0; i_s<N; ++i_s) //site index
  {
    hiprandGenerate(gen,ran,3);
    Jx[i_s] = ran[0]&1;
    Jy[i_s] = ran[1]&1;
    Jz[i_s] = ran[2]&1;
  }

  //copy coupling constants to lattice
  for (int xa = 0; xa<L; ++xa) //advanced x index
  {
    int xr = (xa+L-1)%L; //retarded x index
    for (int ya = 0; ya<L; ++ya) //advanced y index
    {
      int yr = (ya+L-1)%L; //retarded y index
      for (int za = 0; za<L; ++za) //advanced z index
      {
        int zr = (za+L-1)%L; //retarded z index
        uint32_t J = //site's coupling constants
          (MASKSJ<<0)*Jx[L*L*za+L*ya+xr]|
          (MASKSJ<<1)*Jx[L*L*za+L*ya+xa]|
          (MASKSJ<<2)*Jy[L*L*za+L*yr+xa]|
          (MASKSJ<<3)*Jy[L*L*za+L*ya+xa]|
          (MASKSJ<<4)*Jz[L*L*zr+L*ya+xa]|
          (MASKSJ<<5)*Jz[L*L*za+L*ya+xa];
        int i_s = L*L*za+L*ya+xa; //site index
        lattice_h[i_s] = J|(lattice_h[i_s]&MASKAS);
      }
    }
  }
}

//initialize lattice array
void eamsim::init_lattice()
{
  //initialize host PRNG
  hiprandGenerator_t gen; //host PRNG
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //initialize every lattice in the host
  for (int i_l = 0; i_l<NL; ++i_l) //lattice index
  {
    if ((i_l%NCP)==0) //initialize lattice coupling constants
    {
      init_coupling_constants(gen,&lattice_h[N*i_l]);
    }
    else //use the same coupling constants for adjacent lattices
    {
      cuda_check(hipMemcpy(&lattice_h[N*i_l],&lattice_h[N*(i_l-1)],
        N*sizeof(uint32_t),hipMemcpyHostToHost));
    }

    //initialize lattice multispins
    init_multispins(gen,&lattice_h[N*i_l]);
  }

  //copy lattice host array to device
  cuda_check(hipMemcpy(lattice,lattice_h,N*NL*sizeof(uint32_t),
    hipMemcpyHostToDevice));

  //record success message
  logger::record("lattice array initialized");
}

//save state to binary file
void eamsim::save_checkpoint(std::ofstream &bin_out_f) //binary output file
{
  //write lattice array to binary file
  bin_out_f.write(reinterpret_cast<char *>(lattice_h),N*NL*sizeof(uint32_t));

  //record success message
  logger::record("checkpoint saved");
}

//load state from binary file
void eamsim::load_checkpoint(std::ifstream &bin_inp_f) //binary input file
{
  //read lattice array from binary file
  bin_inp_f.read(reinterpret_cast<char *>(lattice_h),N*NL*sizeof(uint32_t));

  //copy lattice array to device
  cuda_check(hipMemcpy(lattice,lattice_h,N*NL*sizeof(uint32_t),
    hipMemcpyHostToDevice));

  //record success message
  logger::record("checkpoint loaded");
}

//run whole simulation
void eamsim::run_simulation(std::ofstream &bin_out_f) //binary output file
{
  //declare and initialize execution time
  float t_e = clock(); //execution time

  //copy lattice array to shuffled lattice array
  cuda_check(hipMemcpy(slattice,lattice,N*NL*sizeof(uint32_t),
    hipMemcpyDeviceToDevice));

  //run whole simulation
  for (int step = 0; step<SPFILE; step += SBMEAS) //Monte Carlo step index
  {
    //show simulation progress
    logger::show_prog_pc(100.0*step/SPFILE);

    //run simulation section between measurements
    run_simulation_section<<<NL,CBDIM>>>(slattice,obs,repib,H,prngs);

    //rearrange lattice temperature replicas
    rearrange<<<NL,NTPB>>>(lattice,repib,slattice);

    //compute overlap
    compute_q<<<NDIS,NTPB>>>(obs,lattice);

    //copy observables array to host
    cuda_check(hipMemcpy(obs_h,obs,NREP*NDIS*sizeof(obs_s),
      hipMemcpyDeviceToHost));

    //write observables to binary file
    bin_out_f.write(reinterpret_cast<char *>(obs_h),NREP*NDIS*sizeof(obs_s));
  }

  //copy lattice array to host
  cuda_check(hipMemcpy(lattice_h,lattice,N*NL*sizeof(uint32_t),
    hipMemcpyDeviceToHost));

  //compute and record execution time
  t_e = (clock()-t_e)/CLOCKS_PER_SEC;
  t_e *= (1e12/SPFILE)/(N*NL*NREP);
  logger::record("t_e = "+cnfs(t_e,5,'0',2)+" ps");

  //record success message
  logger::record("simulation ended");
}

//initialize replica index-beta array
void eamsim::init_repib()
{
  //declare auxiliary variables
  const float max_beta = 2.0; //maximum beta
  const float bratio = pow(2.0,-4/(NREP-1.0)); //beta ratio

  //initialize replica index-beta host array
  for (int i_l = 0; i_l<NL; ++i_l) //lattice index
  {
    for (int i_r = 0; i_r<NREP; ++i_r) //replica index
    {
      repib_h[NREP*i_l+i_r].idx = i_r;
      repib_h[NREP*i_l+i_r].beta = pow(bratio,i_r)*max_beta;
    }
  }

  //copy replica index-beta host array to device
  cuda_check(hipMemcpy(repib,repib_h,NREP*NL*sizeof(ib_s),
    hipMemcpyHostToDevice));
}
