//Includes

#include "eamsim.cuh" //EA model simulation

#include <time.h> //time utilities library

#include <hiprand/hiprand_kernel.h> //cuRAND device functions

//Namespace

namespace mmc //Marco Mendívil Carboni
{

//Constants

//Aliases

using prng = hiprandStatePhilox4_32_10; //PRNG type

//Enumerations

//Device Functions

//Host Functions

//EA model simulation constructor
eamsim::eamsim(parmap &par) //parameters
{
  //check parameters
  std::string msg = "hello :D"; //message
  logger::record(msg);

  //allocate device memory

  //allocate host memory

  //initialize PRNG
}

//EA model simulation destructor
eamsim::~eamsim()
{
  //deallocate device memory

  //deallocate host memory
}

} //namespace mmc
