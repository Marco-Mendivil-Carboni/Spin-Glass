#include "hip/hip_runtime.h"
//Includes

#include "eamsim.cuh" //EA model simulation

#include <time.h> //time utilities library

#include <hiprand/hiprand_kernel.h> //cuRAND device functions

//Constants

static constexpr uint MCSBS = 32; //Monte Carlo steps between shuffles

static constexpr uint NTPB = 256; //number of threads per block
static constexpr uint NBPG = NDIS; //number of blocks per grid

//Aliases

using prng = hiprandStatePhilox4_32_10; //PRNG type

//Device Functions

//sequential addressing reduction
__device__ void reduction(
  float *a,
  short s_a[NREP][NTPB], //shared vec
  const uint i_bt) //block thread index
{

}

//Global Functions

//initialize PRNG state array
__global__ void init_prng(
  void *vprngs, //void PRNG state array
  uint pseed) //PRNG seed
{
  //calculate grid thread index
  const uint i_gt = blockDim.x*blockIdx.x+threadIdx.x; //grid thread index

  //initialize PRNG state
  prng *prngs = static_cast<prng *>(vprngs); //PRNG state array
  hiprand_init(pseed,i_gt,0,&prngs[i_gt]);
}

//rearrange lattice temperature replicas
__global__ void rearrange(
  uint *lattice, //lattice array
  ibeta *rbeta, //replica beta array
  uint *slattice) //shuffled lattice array
{
  //calculate indexes
  const uint i_bt = threadIdx.x; //block thread index
  const uint i_gb = blockIdx.x; //grid block index

  //declare auxiliary variables
  uint smspin; //shuffled multispin
  uint rmspin; //rearranged multispin
  __shared__ uint s_rbeta_idx[NREP]; //shared replica beta index

  //write shared replica beta index
  if (i_bt<NREP){ s_rbeta_idx[i_bt] = rbeta[NREP*i_gb+i_bt].idx;}
  __syncthreads();

  //update lattice array
  for (uint i_s = i_bt; i_s<N; i_s += NTPB) //site index
  {
    smspin = slattice[N*i_gb+i_s];
    rmspin = 0;
    for (uint i_b = 0; i_b<NREP; ++i_b) //beta index
    {
	    rmspin |= ((smspin>>i_b)&1)<<s_rbeta_idx[i_b];
    }
    lattice[N*i_gb+i_s] = (lattice[N*i_gb+i_s]&MASKAJ)|rmspin;
  }
}

//Host Functions

//EA model simulation constructor
eamsim::eamsim(float beta) //inverse temperature
  : eamdat()
  , beta {beta}
{
  //check parameters
  if (!(0.125<=beta&&beta<=8.0)){ throw error("beta out of range");}
  logger::record("beta = "+cnfs(beta,5,'0',3));

  //allocate device memory
  cuda_check(hipMalloc(&rbeta,NREP*NDIS*sizeof(ibeta)));
  cuda_check(hipMalloc(&vprngs,NTPB*NBPG*sizeof(prng)));
  cuda_check(hipMalloc(&slattice,N*NDIS*sizeof(uint)));

  //allocate host memory
  cuda_check(hipHostMalloc(&rbeta_h,NREP*NDIS*sizeof(ibeta)));

  //initialize replica beta array
  init_rbeta();

  //initialize PRNG state array
  init_prng<<<NTPB,NBPG>>>(vprngs,time(nullptr));

  //record success message
  logger::record("eamsim initialized");
}

//EA model simulation destructor
eamsim::~eamsim()
{
  //deallocate device memory
  cuda_check(hipFree(rbeta));
  cuda_check(hipFree(vprngs));
  cuda_check(hipFree(slattice));

  //deallocate host memory
  cuda_check(hipHostFree(rbeta_h));
}

//initialize lattice multispins
void init_multispins(
  hiprandGenerator_t gen, //host PRNG
  uint *lattice_h) //lattice host array
{
  //declare auxiliary variables
  uint ranmspin; //random multispin

  //set random lattice multispins
  for (uint i_s = 0; i_s<N; ++i_s) //site index
  {
    hiprandGenerate(gen,&ranmspin,1);
    lattice_h[i_s] = (lattice_h[i_s]&MASKAJ)|ranmspin&MASKAS;
  }
}

//initialize lattice coupling constants
void init_coupling_constants(
  hiprandGenerator_t gen, //host PRNG
  uint *lattice_h) //lattice host array
{
  //declare auxiliary variables
  uint ran[3]; //random numbers
  uint Jx[N]; //x coupling constants
  uint Jy[N]; //y coupling constants
  uint Jz[N]; //z coupling constants

  //choose random coupling constants
  for (uint i_s = 0; i_s<N; ++i_s) //site index
  {
    hiprandGenerate(gen,ran,3);
    Jx[i_s] = ran[0]&1;
    Jy[i_s] = ran[1]&1;
    Jz[i_s] = ran[2]&1;
  }

  //copy coupling constants to lattice
  for (uint xa = 0; xa<L; ++xa) //advanced x index
  {
    uint xr = (xa+L-1)%L; //retarded x index
    for (uint ya = 0; ya<L; ++ya) //advanced y index
    {
      uint yr = (ya+L-1)%L; //retarded y index
      for (uint za = 0; za<L; ++za) //advanced z index
      {
	      uint zr = (za+L-1)%L; //retarded z index
	      uint J = //site's coupling constants
	        (MASKSJ<<0)*Jx[L*L*za+L*ya+xr]|
	        (MASKSJ<<1)*Jx[L*L*za+L*ya+xa]|
	        (MASKSJ<<2)*Jy[L*L*za+L*yr+xa]|
	        (MASKSJ<<3)*Jy[L*L*za+L*ya+xa]|
	        (MASKSJ<<4)*Jz[L*L*zr+L*ya+xa]|
	        (MASKSJ<<5)*Jz[L*L*za+L*ya+xa];
        uint i_s = L*L*za+L*ya+xa; //site index
        lattice_h[i_s] = J|(lattice_h[i_s]&MASKAS);
      }
    }
  }
}

//initialize lattice array
void eamsim::init_lattice()
{
  //initialize host PRNG
  hiprandGenerator_t gen; //host PRNG
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //initialize every lattice in the host
  for (uint i_l = 0; i_l<NDIS; ++i_l) //lattice index
  {
    init_multispins(gen,&lattice_h[N*i_l]);
    init_coupling_constants(gen,&lattice_h[N*i_l]);
  }

  //copy lattice host array to device
  cuda_check(hipMemcpy(lattice,lattice_h,N*NDIS*sizeof(uint),
    hipMemcpyHostToDevice));

  //record success message
  logger::record("lattice array initialized");
}

//run Monte Carlo simulation
void eamsim::run_MC_simulation(std::ofstream &bin_out_f) //binary output file
{
  //copy lattice array to shuffled lattice array
  cuda_check(hipMemcpy(slattice,lattice,N*NDIS*sizeof(uint),
    hipMemcpyDeviceToDevice));

  //Monte Carlo steps...

  //rearrange lattice temperature replicas
  rearrange<<<NTPB,NBPG>>>(lattice,rbeta,slattice);

  //copy lattice array to host
  cuda_check(hipMemcpy(lattice_h,lattice,N*NDIS*sizeof(uint),
    hipMemcpyDeviceToHost));

  //write state to binary file
  write_state(bin_out_f);

  //record success message
  logger::record("simulation ended");
}

//initialize replica beta array
void eamsim::init_rbeta()
{
  //declare auxiliary variables
  const float bratio = pow(2.0,4/(NREP-1.0)); //beta ratio

  //initialize replica beta host array
  for (uint i_l = 0; i_l<NDIS; ++i_l) //lattice index
  {
    for (uint i_b = 0; i_b<NREP; ++i_b) //beta index
    {
      rbeta_h[NREP*i_l+i_b].idx = i_b;
      rbeta_h[NREP*i_l+i_b].beta = pow(bratio,i_b)*beta;
    }
  }

  //copy replica beta host array to device
  cuda_check(hipMemcpy(rbeta,rbeta_h,NREP*NDIS*sizeof(ibeta),
    hipMemcpyHostToDevice));
}
