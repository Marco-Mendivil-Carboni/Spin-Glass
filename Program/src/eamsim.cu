#include "hip/hip_runtime.h"
//Includes

#include "eamsim.cuh" //EA model simulation

#include <time.h> //time utilities library

#include <hiprand/hiprand_kernel.h> //cuRAND device functions

//Constants

static constexpr float H = 0.0; //external magnetic field

static constexpr uint NPROB = 14; //number of possible probabilities
static constexpr uint PTABW = 16; //probability lookup table width

static constexpr uint MCSBS = 32; //Monte Carlo steps between shuffles

static constexpr uint NTPB = 256; //number of threads per block
static constexpr dim3 CBDIM = {L/2,L,2}; //checkerboard block dimensions
static constexpr uint NBPG = NDIS; //number of blocks per grid

//Aliases

using prng = hiprandStatePhilox4_32_10; //PRNG type

//Device Functions

//initialize probability lookup table
inline __device__ void init_prob(
  uint s_prob[NREP][PTABW], //shared probability lookup table
  const uint i_bt) //block thread index
{
  //initialize all entries to 1
  if (i_bt<PTABW)
  {
    for (uint i_b = 0; i_b<NREP; ++i_b) //beta index
    {
      s_prob[i_b][i_bt] = UINT_MAX;
    }
  }
  __syncthreads();
}

//compute probability lookup table
inline __device__ void compute_prob(
  uint s_prob[NREP][PTABW], //shared probability lookup table
  float *s_rep_beta, //shared replica beta array
  const uint i_bt) //block thread index
{
  //compute all possible probabilities
  if (i_bt<NPROB)
  {
    for (uint i_b = 0; i_b<NREP; ++i_b) //beta index
    {
      float energy = i_bt-6+H-((1+2*H)*(i_bt&1)); //spin energy
      s_prob[i_b][i_bt] = expf(s_rep_beta[i_b]*2*energy)*UINT_MAX;
    }
  }
  __syncthreads();
}

//shuffle lattice temperature replicas
inline __device__ void shuffle(
  uint *s_rep_idx, //shared replica index array
  float *s_rep_beta, //shared replica beta array
  float *tot_energy, //total energy array
  void *vprngs, //void PRNG state array
  const uint i_bt, //block thread index
  const uint i_gt, //grid thread index
  bool mode) //shuffle mode
{
  //declare auxiliary variables
  uint i_0; //1st array index
  uint i_1; //2nd array index
  uint max_i_bt; //maximum block thread index
  __shared__ uint s_rai[NREP]; //shared rearranged array index array

  //write shared rearranged array index array
  if (i_bt<NREP){ s_rai[s_rep_idx[i_bt]] = i_bt;}
  __syncthreads ();

  if (!mode) //consider even pairs of temperature replicas
  {
    i_0 = s_rai[(i_bt<<1)+0]; i_1 = s_rai[(i_bt<<1)+1]; max_i_bt = NREP/2;
  }
  else //consider odd pairs of temperature replicas
  {
    i_0 = s_rai[(i_bt<<1)+1]; i_1 = s_rai[(i_bt<<1)+2]; max_i_bt = NREP/2-1;
  }

  if (i_bt<max_i_bt) //shuffle pair of temperature replicas
  {
    //generate random number
    prng *prngs = static_cast<prng *>(vprngs); //PRNG state array
    float ran = hiprand_uniform(&prngs[i_gt]); //random number in (0,1]

    //compute shuffle probability
    float beta_diff = s_rep_beta[i_0]-s_rep_beta[i_1]; //beta difference
    float energy_diff = tot_energy[i_0]-tot_energy[i_1]; //energy difference
    float prob = expf(beta_diff*energy_diff); //shuffle probability

    if (ran<prob) //accept shuffle
    {
      uint tmp_idx = s_rep_idx[i_0]; //temporary index
      s_rep_idx[i_0] = s_rep_idx[i_1]; s_rep_idx[i_1] = tmp_idx;
      float tmp_beta = s_rep_beta[i_0]; //temporary beta
      s_rep_beta[i_0] = s_rep_beta[i_1]; s_rep_beta[i_1] = tmp_beta;
    }
  }
  __syncthreads();
}

//perform skewed sequential sum reduction
inline __device__ void sum_reduce(
  float *tot_energy, //total energy array
  short aux_energy[NREP][NTPB], //auxiliary energy array
  const uint i_bt) //block thread index
{
  //sum auxiliary energies for each temperature replica
  if (i_bt<NREP)
  {
    int sum = 0; //sum of energies
    for (uint i_sl = 0; i_sl<NTPB; ++i_sl) //skewed loop index
    {
      sum += aux_energy[i_bt][(i_sl+i_bt)%NTPB];
    }
    tot_energy[i_bt] = sum;
  }
  __syncthreads();
}

//perform Monte Carlo steps
inline __device__ void perform_MC_steps(
  uint *slattice, //shuffled lattice array
  float *s_rep_beta, //shared replica beta array
  void *vprngs, //void PRNG state array
  int iter) //...
{
  //calculate indexes
  const uint i_gb = blockIdx.x; //grid block index
  const uint i_bt = //block thread index
    CBDIM.x*CBDIM.y*threadIdx.z+CBDIM.x*threadIdx.y+threadIdx.x;
  const uint i_gt = CBDIM.x*CBDIM.y*CBDIM.z*i_gb+i_bt; //grid thread index

  //declare auxiliary variables
  __shared__ uint s_prob[NREP][PTABW]; //shared probability lookup table
  __shared__ uint l[L][L][L]; //shared single lattice

  //initilize probability lookup table
  init_prob(s_prob,i_bt);

  //compute probability lookup table
  compute_prob(s_prob,s_rep_beta,i_bt);

  // index for read/write global memory
  int xx = (L/2*(threadIdx.y&1))+threadIdx.x;
  int yy = (L/2*(threadIdx.z&1))+(threadIdx.y>>1);

  // import lattice scratchpad
  for (int z_offset = 0; z_offset < L; z_offset += (CBDIM.z>>1))
  {
    int zz = z_offset + (threadIdx.z >> 1);
    l[zz][yy][xx] = slattice[N*i_gb + L * L * z_offset + i_bt];
  }
  __syncthreads();

  for (int i = 0; i < iter; i++)
  {
    //two phases update
    for (int run = 0; run < 2; run++)
    {
  	  int x0 = (threadIdx.z & 1) ^ (threadIdx.y & 1) ^ run;	// initial x

  	  int x = (threadIdx.x << 1) + x0;
  	  int xa = (x + L - 1) % L; //retarded...
  	  int xb = (x + 1) % L; //advanced...

      int y = threadIdx.y;
      int ya = (y + L - 1) % L;
      int yb = (y + 1) % L;

  	  for (int z_offset = 0; z_offset < L; z_offset += CBDIM.z)
      {
  	    int z = z_offset + threadIdx.z;
  	    int za = (z + L - 1) % L;
  	    int zb = (z + 1) % L;

        //...
    	  uint c =  l[z][y][x]; //center
  	    uint n0 = l[z][y][xa]; //left
  	    uint n1 = l[z][y][xb]; //right
  	    uint n2 = l[z][ya][x]; //up
  	    uint n3 = l[z][yb][x]; //down
  	    uint n4 = l[za][y][x]; //front
  	    uint n5 = l[zb][y][x]; //back

        //...
  	    n0 = (MASKAB*((c>>(SHIFTSJ+0))&1))^n0^c;
  	    n1 = (MASKAB*((c>>(SHIFTSJ+1))&1))^n1^c;
  	    n2 = (MASKAB*((c>>(SHIFTSJ+2))&1))^n2^c;
  	    n3 = (MASKAB*((c>>(SHIFTSJ+3))&1))^n3^c;
  	    n4 = (MASKAB*((c>>(SHIFTSJ+4))&1))^n4^c;
  	    n5 = (MASKAB*((c>>(SHIFTSJ+5))&1))^n5^c;

  	    for (int s = 0; s < NSPS; s++)
        {
          //...
  	      uint e = //...
  	        ((n0>>s)&MASKSS)+
  	        ((n1>>s)&MASKSS)+
  	        ((n2>>s)&MASKSS)+
  	        ((n3>>s)&MASKSS)+
  	        ((n4>>s)&MASKSS)+
  	        ((n5>>s)&MASKSS);
  	      e = (e << 1) + ((c >> s) & MASKSS);

          //...
  	      uint flip = 0; //...
    	    for (int shift = 0; shift < SHIFTMS; shift += NSPS)
          {
  	        uint val = s_prob[shift+s][(e>>shift)&MASKES]; //...
            prng *prngs = static_cast<prng *>(vprngs); //PRNG state array
  	        uint myrand = hiprand(&prngs[i_gt]);	//...
  	        flip |= (myrand<val)<<shift;
  	      }

          //flip spins
  	      c ^= (flip << s);
  	    }

        //save new spins
  	    l[z][y][x] = c;
  	  }
  	  __syncthreads();
    }
  }

  //...
  for (int z_offset = 0; z_offset < L; z_offset += (CBDIM.z >> 1))
  {
    int zz = z_offset + (threadIdx.z >> 1);
    slattice[N*i_gb+L*L*z_offset+i_bt] = l[zz][yy][xx];
  }
  __syncthreads();
}

//stencil_swap...

//Global Functions

//initialize PRNG state array
__global__ void init_prng(
  void *vprngs, //void PRNG state array
  uint pseed) //PRNG seed
{
  //calculate grid thread index
  const uint i_gt = NTPB*blockIdx.x+threadIdx.x; //grid thread index

  //initialize PRNG state
  prng *prngs = static_cast<prng *>(vprngs); //PRNG state array
  hiprand_init(pseed,i_gt,0,&prngs[i_gt]);
}

//kernel_warmup...

//kernel_swap...

//rearrange lattice temperature replicas
__global__ void rearrange(
  uint *lattice, //lattice array
  ib_s *repib, //replica index-beta array
  uint *slattice) //shuffled lattice array
{
  //calculate indexes
  const uint i_gb = blockIdx.x; //grid block index
  const uint i_bt = threadIdx.x; //block thread index

  //declare auxiliary variables
  uint smspin; //shuffled multispin
  uint rmspin; //rearranged multispin
  __shared__ uint s_rep_idx[NREP]; //shared replica index array

  //write shared replica index array
  if (i_bt<NREP){ s_rep_idx[i_bt] = repib[NREP*i_gb+i_bt].idx;}
  __syncthreads();

  //update lattice array
  for (uint i_s = i_bt; i_s<N; i_s += NTPB) //site index
  {
    smspin = slattice[N*i_gb+i_s];
    rmspin = 0;
    for (uint i_b = 0; i_b<NREP; ++i_b) //beta index
    {
	    rmspin |= ((smspin>>i_b)&1)<<s_rep_idx[i_b];
    }
    lattice[N*i_gb+i_s] = (lattice[N*i_gb+i_s]&MASKAJ)|rmspin;
  }
}

//Host Functions

//EA model simulation constructor
eamsim::eamsim(float beta) //inverse temperature
  : eamdat()
  , beta {beta}
{
  //check parameters
  if (!(0.125<=beta&&beta<=8.0)){ throw error("beta out of range");}
  logger::record("beta = "+cnfs(beta,5,'0',3));

  //allocate device memory
  cuda_check(hipMalloc(&repib,NREP*NDIS*sizeof(ib_s)));
  cuda_check(hipMalloc(&vprngs,NTPB*NBPG*sizeof(prng)));
  cuda_check(hipMalloc(&slattice,N*NDIS*sizeof(uint)));

  //allocate host memory
  cuda_check(hipHostMalloc(&repib_h,NREP*NDIS*sizeof(ib_s)));

  //initialize replica index-beta array
  init_repib();

  //initialize PRNG state array
  init_prng<<<NTPB,NBPG>>>(vprngs,time(nullptr));

  //record success message
  logger::record("eamsim initialized");
}

//EA model simulation destructor
eamsim::~eamsim()
{
  //deallocate device memory
  cuda_check(hipFree(repib));
  cuda_check(hipFree(vprngs));
  cuda_check(hipFree(slattice));

  //deallocate host memory
  cuda_check(hipHostFree(repib_h));
}

//initialize lattice multispins
void init_multispins(
  hiprandGenerator_t gen, //host PRNG
  uint *lattice_h) //lattice host array
{
  //declare auxiliary variables
  uint ranmspin; //random multispin

  //set random lattice multispins
  for (uint i_s = 0; i_s<N; ++i_s) //site index
  {
    hiprandGenerate(gen,&ranmspin,1);
    lattice_h[i_s] = (lattice_h[i_s]&MASKAJ)|(ranmspin&MASKAS);
  }
}

//initialize lattice coupling constants
void init_coupling_constants(
  hiprandGenerator_t gen, //host PRNG
  uint *lattice_h) //lattice host array
{
  //declare auxiliary variables
  uint ran[3]; //random numbers
  uint Jx[N]; //x coupling constants
  uint Jy[N]; //y coupling constants
  uint Jz[N]; //z coupling constants

  //choose random coupling constants
  for (uint i_s = 0; i_s<N; ++i_s) //site index
  {
    hiprandGenerate(gen,ran,3);
    Jx[i_s] = ran[0]&1;
    Jy[i_s] = ran[1]&1;
    Jz[i_s] = ran[2]&1;
  }

  //copy coupling constants to lattice
  for (uint xa = 0; xa<L; ++xa) //advanced x index
  {
    uint xr = (xa+L-1)%L; //retarded x index
    for (uint ya = 0; ya<L; ++ya) //advanced y index
    {
      uint yr = (ya+L-1)%L; //retarded y index
      for (uint za = 0; za<L; ++za) //advanced z index
      {
	      uint zr = (za+L-1)%L; //retarded z index
	      uint J = //site's coupling constants
	        (MASKSJ<<0)*Jx[L*L*za+L*ya+xr]|
	        (MASKSJ<<1)*Jx[L*L*za+L*ya+xa]|
	        (MASKSJ<<2)*Jy[L*L*za+L*yr+xa]|
	        (MASKSJ<<3)*Jy[L*L*za+L*ya+xa]|
	        (MASKSJ<<4)*Jz[L*L*zr+L*ya+xa]|
	        (MASKSJ<<5)*Jz[L*L*za+L*ya+xa];
        uint i_s = L*L*za+L*ya+xa; //site index
        lattice_h[i_s] = J|(lattice_h[i_s]&MASKAS);
      }
    }
  }
}

//initialize lattice array
void eamsim::init_lattice()
{
  //initialize host PRNG
  hiprandGenerator_t gen; //host PRNG
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //initialize every lattice in the host
  for (uint i_l = 0; i_l<NDIS; ++i_l) //lattice index
  {
    init_multispins(gen,&lattice_h[N*i_l]);
    init_coupling_constants(gen,&lattice_h[N*i_l]);
  }

  //copy lattice host array to device
  cuda_check(hipMemcpy(lattice,lattice_h,N*NDIS*sizeof(uint),
    hipMemcpyHostToDevice));

  //record success message
  logger::record("lattice array initialized");
}

//run Monte Carlo simulation
void eamsim::run_MC_simulation(std::ofstream &bin_out_f) //binary output file
{
  //copy lattice array to shuffled lattice array
  cuda_check(hipMemcpy(slattice,lattice,N*NDIS*sizeof(uint),
    hipMemcpyDeviceToDevice));

  //Monte Carlo steps...

  //rearrange lattice temperature replicas
  rearrange<<<NTPB,NBPG>>>(lattice,repib,slattice);

  //copy lattice array to host
  cuda_check(hipMemcpy(lattice_h,lattice,N*NDIS*sizeof(uint),
    hipMemcpyDeviceToHost));

  //write state to binary file
  write_state(bin_out_f);

  //record success message
  logger::record("simulation ended");
}

//initialize replica index-beta array
void eamsim::init_repib()
{
  //declare auxiliary variables
  const float bratio = pow(2.0,4/(NREP-1.0)); //beta ratio

  //initialize replica index-beta host array
  for (uint i_l = 0; i_l<NDIS; ++i_l) //lattice index
  {
    for (uint i_b = 0; i_b<NREP; ++i_b) //beta index
    {
      repib_h[NREP*i_l+i_b].idx = i_b;
      repib_h[NREP*i_l+i_b].beta = pow(bratio,i_b)*beta;
    }
  }

  //copy replica index-beta host array to device
  cuda_check(hipMemcpy(repib,repib_h,NREP*NDIS*sizeof(ib_s),
    hipMemcpyHostToDevice));
}
