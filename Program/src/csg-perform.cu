#include "hip/hip_runtime.h"
//Includes

#include "eamsim.cuh" //EA model simulation

#include <iostream> //standard input/output stream objects

//Functions

//main function
int main(
  const int argc, //argument count
  const char **argv) //argument vector
{
  //check command-line arguments
  if (argc<2){ std::cout<<"no arguments\n"; return EXIT_FAILURE;}
  if (argc>2){ std::cout<<"extra arguments\n"; return EXIT_FAILURE;}

  //declare auxiliary variables
  const std::string sim_dir = argv[1]; //simulation directory
  std::ifstream inp_f; //input file
  std::ofstream out_f; //output file
  std::string pathstr; //file path string
  std::string pathpat; //file path pattern
  uint i_s_f; //simulation file index

  //create log file in current working directory
  time_t t_s = time(nullptr); //starting time
  pathstr = std::to_string(t_s)+".log";
  mmc::logger::set_file(pathstr);

  //main try block
  try
  {
    //read parameters and initialize simulation
    pathstr = sim_dir+"/adjustable-parameters.dat";
    inp_f.open(pathstr);
    mmc::check_file(inp_f,pathstr);
    mmc::parmap par(inp_f); //parameters
    inp_f.close();
    mmc::eamsim sim(par); //simulation
  }
  catch (const mmc::error &err) //caught error
  {
    //exit program unsuccessfully
    mmc::logger::record(err.what());
    return EXIT_FAILURE;
  }

  //remove log file
  mmc::logger::set_file("/dev/null");
  pathstr = std::to_string(t_s)+".log";
  std::remove(pathstr.c_str());

  //exit program successfully
  return EXIT_SUCCESS;
}
