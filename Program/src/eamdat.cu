//Includes

#include "eamdat.cuh" //EA model data

//Functions

//EA model data constructor
eamdat::eamdat()
{
  //allocate device memory
  cuda_check(hipMalloc(&lattice,N*NL*sizeof(uint32_t)));

  //allocate host memory
  cuda_check(hipHostMalloc(&lattice_h,N*NL*sizeof(uint32_t)));

  //record success message
  logger::record("eamdat initialized");
}

//EA model data destructor
eamdat::~eamdat()
{
  //deallocate device memory
  cuda_check(hipFree(lattice));

  //deallocate host memory
  cuda_check(hipHostFree(lattice_h));
}

//write state to binary file
void eamdat::write_state(std::ofstream &bin_out_f) //binary output file
{
  //write lattice host array
  bin_out_f.write(reinterpret_cast<char *>(lattice_h),N*NL*sizeof(uint32_t));

  //check filestream
  if (bin_out_f.fail()){ throw error("failed to write state");}
}

//read state from binary file
void eamdat::read_state(std::ifstream &bin_inp_f) //binary input file
{
  //read lattice host array
  bin_inp_f.read(reinterpret_cast<char *>(lattice_h),N*NL*sizeof(uint32_t));

  //copy lattice host array to device
  cuda_check(hipMemcpy(lattice,lattice_h,N*NL*sizeof(uint32_t),
    hipMemcpyHostToDevice));

  //check filestream
  if (bin_inp_f.fail()){ throw error("failed to read state");}
}
