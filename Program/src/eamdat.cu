//Includes

#include "eamdat.cuh" //EA model data

//Namespace

namespace mmc //Marco Mendívil Carboni
{

//Functions

//EA model data constructor
eamdat::eamdat()
{
  //allocate device memory
  cuda_check(hipMalloc(&lattice,NDIS*N*sizeof(uint)));

  //allocate host memory
  cuda_check(hipHostMalloc(&lattice_h,NDIS*N*sizeof(uint)));

  //record success message
  std::string msg = "eamdat initialized "; //message
  logger::record(msg);
}

//EA model data destructor
eamdat::~eamdat()
{
  //deallocate device memory
  cuda_check(hipFree(lattice));

  //deallocate host memory
  cuda_check(hipHostFree(lattice_h));
}

//write state to binary file
void eamdat::write_state(std::ofstream &bin_out_f) //binary output file
{
  //write lattice host array
  bin_out_f.write(reinterpret_cast<char *>(lattice_h),NDIS*N*sizeof(uint));

  //check filestream
  if (bin_out_f.fail())
  {
    throw mmc::error("failed to write state to binary file");
  }
}

//read state from binary file
void eamdat::read_state(std::ifstream &bin_inp_f) //binary input file
{
  //read lattice host array
  bin_inp_f.read(reinterpret_cast<char *>(lattice_h),NDIS*N*sizeof(uint));

  //copy lattice host array to device
  cuda_check(hipMemcpy(lattice,lattice_h,NDIS*N*sizeof(uint),
    hipMemcpyHostToDevice));

  //check filestream
  if (bin_inp_f.fail())
  {
    throw mmc::error("failed to read state from binary file");
  }
}

} //namespace mmc
